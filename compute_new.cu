#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "compute.h"
#include "vector.h"
#include "config.h"

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL

__global__ void populate_acceleration(vector3* values, vector3** accel, int local_start, int local_end);

__global__ void compute_pairwise_acceleration(vector3* values, vector3** accel, int local_start, int local_end);

__global__ void sum_rows_from_accel_sum(vector3** accel, int loop_index, int local_start, int local_end);
 
void compute(){
    //int index = blockIdx.x * blockDim.x + threadIdx.x;
    //int stride = blockDim.x * gridDim.x;
	//make an acceleration matrix which is NUMENTITIES squared in size;
	int i;

	int grid_size = 50;
	int blocksize = 50;

    /* Probably need cudamalloc*/
	vector3* values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	vector3** accels=(vector3**)malloc(sizeof(vector3*)*NUMENTITIES);

	hipMalloc(&d_hVel, NUMENTITIES * sizeof(vector3));
	hipMalloc(&d_hPos, NUMENTITIES * sizeof(vector3));

	hipMemcpy(d_hVel, hVel, NUMENTITIES * sizeof(vector3), hipMemcpyHostToDevice);
	hipMemcpy(d_hPos, hPos, NUMENTITIES * sizeof(vector3), hipMemcpyHostToDevice);

	hipMemcpy(d_mass, mass, sizeof(double), hipMemcpyHostToDevice);


	//need to copy hval and hpos onto the GPU
	//copy values to values and accel 

    /* kernel function */

    populate_acceleration<<<grid_size,blocksize>>>(values,accels,0,NUMENTITIES);

	/*for (i=0;i<NUMENTITIES;i++)
		accels[i]=&values[i*NUMENTITIES];*/



	//first compute the pairwise accelerations.  Effect is on the first argument.
	
    compute_pairwise_acceleration<<<grid_size,blocksize>>>(values, accels,0, NUMENTITIES);

    /*for (i=0;i<NUMENTITIES;i++){
		for (j=0;j<NUMENTITIES;j++){
			if (i==j) {
				FILL_VECTOR(accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
				FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
		}
	}*/

	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	for (i=0;i<NUMENTITIES;i++){
        sum_rows_from_accel_sum<<<grid_size,blocksize>>>(accels, i, 0, NUMENTITIES); //new kernel function
		/*for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]=hVel[i][k]*INTERVAL;
		}*/
	}
	free(accels);
	free(values);
}


__global__ void populate_acceleration(vector3* values, vector3** accel, int local_start, int local_end){
		for (int i=local_start;i<local_end;i++)
		accel[i]=&values[i*local_end];
}

__global__ void compute_pairwise_acceleration(vector3* values, vector3** accel, int local_start, int local_end){
    for (int i=local_start;i<local_end;i++){
		for (int j=local_start;j<local_end;j++){
			if (i==j) {
				FILL_VECTOR(accel[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (int k=0;k<3;k++) distance[k]=d_hPos[i][k]-d_hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*d_mass[j]/magnitude_sq;
				FILL_VECTOR(accel[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
		}
	}
}

__global__ void sum_rows_from_accel_sum(vector3** accel, int loop_index, int local_start, int local_end){
		vector3 accel_sum={0,0,0};
		for (int j=local_start;j<local_end;j++){
			for (int k=0;k<3;k++)
				accel_sum[k]+=(accel[loop_index][j][k]);
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (int k=0;k<3;k++){
			d_hVel[loop_index][k]+=accel_sum[k]*INTERVAL;
			d_hPos[loop_index][k]=d_hVel[loop_index][k]*INTERVAL;
		}
}